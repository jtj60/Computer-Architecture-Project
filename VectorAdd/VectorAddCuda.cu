#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000
#define THREADS 1
#define CORES 1
__global__ void add( int *a, int *b, int *c ) {
	int tid = threadIdx.x;	// handle date at index
	c[tid] = a[tid] + b[tid];
	__syncthreads();
}
int main( void ) {
	int a[N], b[N], c[N];
	
	int *dA, *dB, *dC;
	
	//allocate memory to GPU
	hipMalloc(	(void**)&dA, N * sizeof(int));
	hipMalloc(	(void**)&dB, N * sizeof(int));
	hipMalloc(	(void**)&dC, N * sizeof(int));
	
	// fill the arrays 'a' and 'b' on the CPU
	for (int i=0; i<N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	//copy arrays a and b to GPU
	hipMemcpy(dA,a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dB,b, N*sizeof(int), hipMemcpyHostToDevice);
	
	add<<<CORES,N>>>( dA, dB, dC );
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy array c from GPU to CPU
	hipMemcpy(c, dC, N*sizeof(int), hipMemcpyDeviceToHost);
	
	// display the results
	
	for (int i=0; i<N; i++) {
		printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}
	
	//free memory from GPU
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	
	return 0;
}
