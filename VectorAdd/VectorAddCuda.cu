#include<stdio.h>
#include <hip/hip_runtime.h>

#define N 10
#define THREADS 1
#define CORES 1
__global__ void add( int *a, int *b, int *c ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;	// handle date at index
	
	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid+=1;
	}
}
int main( void ) {
	int a[N], b[N], c[N];
	
	int *dA, *dB, *dC;
	
	//allocate memory to GPU
	hipMalloc(	(void**)&dA, N * sizeof(int));
	hipMalloc(	(void**)&dB, N * sizeof(int));
	hipMalloc(	(void**)&dC, N * sizeof(int));
	
	// fill the arrays 'a' and 'b' on the CPU
	for (int i=0; i<N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	//copy arrays a and b to GPU
	hipMemcpy(dA,a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dB,b, N*sizeof(int), hipMemcpyHostToDevice);
	
	add<<<CORES,THREADS>>>( dA, dB, dC );
	
	//copy array c from GPU to CPU
	hipMemcpy(c, dC, N*sizeof(int), hipMemcpyDeviceToHost);
	
	// display the results
	
	for (int i=0; i<N; i++) {
		printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}
	
	//free memory from GPU
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	
	return 0;
}
