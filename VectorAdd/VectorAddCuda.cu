#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE 10000
#define THREADS 1
#define CORES 1
#define TILE_SIZE 32
__global__ void add( int *a, int *b, int *c ) {
	int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
	int tid = 	(blockId * blockDim.x) + threadIdx.x; // handle date at index
	if(tid < SIZE)
		c[tid] = a[tid] + b[tid];
	__syncthreads();
}
int main( void ) {
	int a[SIZE], b[SIZE], c[SIZE];
	
	int *dA, *dB, *dC;
	
	//allocate memory to GPU
	hipMalloc(	(void**)&dA, SIZE * sizeof(int));
	hipMalloc(	(void**)&dB, SIZE * sizeof(int));
	hipMalloc(	(void**)&dC, SIZE * sizeof(int));
	
	// fill the arrays 'a' and 'b' on the CPU
	for (int i=0; i<SIZE; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	
	//copy arrays a and b to GPU
	hipMemcpy(dA,a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dB,b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	
	dim3 dimGrid(SIZE/TILE_SIZE,SIZE/TILE_SIZE, 1);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
	
	add<<<dimGrid,dimBlock>>>( dA, dB, dC );
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy array c from GPU to CPU
	hipMemcpy(c, dC, SIZE*sizeof(int), hipMemcpyDeviceToHost);
	
	// display the results
	
	for (int i=0; i<SIZE; i++) {
		printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}
	
	//free memory from GPU
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	
	return 0;
}
