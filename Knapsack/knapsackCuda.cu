#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKS 1
#define THREADS 1
#define SIZE	1000	//MAX array size
#define GLOBAL_W	50
#define TILE_SIZE 100
__global__ void knapsack(int *val,int *wt,int *result) {
	
	int K[SIZE+1][GLOBAL_W+1];
	
	//FOR LOOP
	for(int i=0; i <= SIZE;i++) {
		for(int w=0; w<=GLOBAL_W;w++) {
			if(i == 0 || w == 0)
				K[i][w] = 0;
			else if(wt[i -1] <= w) {
				int a,b;
				a = val[i-1] + K[i-1][w - wt[i - 1]];
				b = K[i-1][w];
				
				K[i][w] = (a > b) ? a : b;
			}
			else
				K[i][w] = K[i -1][w];
		}
	}
	*result = K[SIZE][GLOBAL_W];
}

int main() {
	int val[SIZE];
	int wt[SIZE];
	int result = 0;
	
	int *dVal, *dWt,*dRes;
	
	//Initialize  Array
	for (int i = 0; i<SIZE; i++){
		val[i] = rand() % 100 +1;
		wt[i] = rand() % 100 +1;
	}
	
	//allocate GPU memory
	hipMalloc(&dVal,(sizeof(val)/sizeof(val[0])) * sizeof(int));
	hipMalloc(&dWt,(sizeof(wt)/sizeof(wt[0])) * sizeof(int));
	hipMalloc(&dRes,sizeof(int));
	
	//copy values from cpu to gpu
	hipMemcpy(dVal, &val, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dWt, &wt, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dRes, &result, sizeof(int), hipMemcpyHostToDevice);
	
	/*Define grid and block dimensions
	dim3 dimGrid(SIZE/TILE_SIZE,SIZE/TILE_SIZE, 1);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
	*/
	
	//call kernel function
	knapsack<<<BLOCKS,THREADS>>>(dVal,dWt,dRes);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy result from gpu to cpu
	hipMemcpy(&result,dRes, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Result: %d\n", result);
	
	//free gpu memory
	hipFree(dVal);
	hipFree(dWt);
	hipFree(dRes);
	return 0;
}
