#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKS 1
#define THREADS 1
#define SIZE	1000	//MAX array size
#define GLOBAL_W	50

__global__ void knapsack(int *val, int *wt,int *result) {
	int K[SIZE + 1][GLOBAL_W + 1];
	
	//FOR LOOP
	for(int i =0;i <= SIZE;i++) {
		for(int w = 0;w <= GLOBAL_W;w++){
			
			if(i == 0 || w == 0)
				K[i][w] = 0;
			else if(wt[i -1] <= w) {
				int a,b;
				a = val[i-1] + K[i-1][w - wt[i - 1]];
				b = K[i-1][w];
				
				K[i][w] = (a > b) ? a : b;
			}
			else
				K[i][w] = K[i -1][w];
		}
	}
	*result = K[SIZE][GLOBAL_W];
}

int main() {
	int val[SIZE];
	int wt[SIZE];
	int result = 0;
	
	int *dVal, *dWt,*dRes;
	
	//Initialize  Array
	for (int i = 0; i<SIZE; i++){
		val[i] = rand() % 100 +1;
		wt[i] = rand() % 100 +1;
	}
	
	//allocate GPU memory
	hipMalloc(&dVal,(sizeof(val)/sizeof(val[0])) * sizeof(int));
	hipMalloc(&dWt,(sizeof(wt)/sizeof(wt[0])) * sizeof(int));
	hipMalloc(&dRes,sizeof(int));
	
	//copy values from cpu to gpu
	hipMemcpy(dVal, &val, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dWt, &wt, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dRes, &result, sizeof(int), hipMemcpyHostToDevice);
	
	
	//call kernel function
	knapsack<<<BLOCKS,THREADS>>>(dVal,dWt,dRes);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy result from gpu to cpu
	hipMemcpy(&result,dRes, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Result: %d\n", result);
	
	//free gpu memory
	hipFree(dVal);
	hipFree(dWt);
	hipFree(dRes);
	return 0;
}
