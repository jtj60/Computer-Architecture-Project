#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKS 1
#define THREADS 1

__global__ void knapsack(int *W, int *val, int *wt, int *n, int *result) {
	/*base
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int w = blockIdx.x*blockDim.x + threadIdx.x;
	
	int K[sizeof(n) + 1][sizeof(W) + 1];
	
	if(i<= *n) {
		if( w<= *W){
			if(i==0 || w==0)
				K[i][w] = 0;
				
			else if(wt[i -1] <= w) {
				int a,b;
				a = val[i-1] + K[i-1][w- wt[i-1]];
				b = K[i-1][w];
				
				K[i][w] = (a>b) ? a : b;
			}
			else
				K[i][w] = K[i -1][w];
		}
	}
	
	*result = K[*n][*W];*/
	
	int K[sizeof(W) + 1][sizeof(n) + 1];
	
	for(int i = 0; i <= (*n); i++) {
		for(int w = 0; w <= (*W); w++){
			
			if(i == 0 || w == 0)
				K[i][w] = 0;
			else if(wt[i -1] <= w) {
				int a,b;
				a = val[i-1] + K[i-1][w - wt[i - 1]];
				b = K[i-1][w];
				
				K[i][w] = (a >= b) ? a : b;
			}
			else
				K[i][w] = K[i -1][w];
		}
	}
	
	*result = K[*n][*W];
}

int main() {
	int val[] = {60,100,120};
	int wt[] = {10,20,30};
	int W = 50;
	int result = 0;
	int n = sizeof(val)/sizeof(val[0]);
	
	int *dVal, *dWt, *dW, *dRes, *dN;
	
	//allocate GPU memory
	hipMalloc(&dVal,sizeof(val) * sizeof(int));
	hipMalloc(&dWt,sizeof(wt) * sizeof(int));
	hipMalloc(&dW,sizeof(int));
	hipMalloc(&dRes,sizeof(int));
	hipMalloc(&dN,sizeof(int));
	
	//copy values from cpu to gpu
	hipMemcpy(dVal, &val, sizeof(val) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dWt, &wt, sizeof(wt) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dW, &W, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dRes, &result, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dW, &W, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dN, &n, sizeof(int), hipMemcpyHostToDevice);
	
	//call kernel function
	knapsack<<<BLOCKS,THREADS>>>(dW,dVal,dWt,dN,dRes);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy result from gpu to cpu
	hipMemcpy(&result,dRes, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Result: %d\n", result);
	
	//free gpu memory
	hipFree(dW);
	hipFree(dVal);
	hipFree(dWt);
	hipFree(dN);
	hipFree(dRes);
	return 0;
}
