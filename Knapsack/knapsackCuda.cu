#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKS 1
#define THREADS 1

#define N	3
#define GLOBAL_W	50

__global__ void knapsack(int *W, int *val, int *wt, int *n, int *result) {
	int K[N + 1][GLOBAL_W + 1];
	
	/*GPU parallel
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int w = blockIdx.y * blockDim.y + threadIdx.y;
	
	while(i <= (*n)) {
		while(w <= (*W)){
			
			if(i == 0 || w == 0)
				K[i][w] = 0;
			else if(wt[i -1] <= w) {
				int a,b;
				a = val[i-1] + K[i-1][w - wt[i - 1]];
				b = K[i-1][w];
				
				K[i][w] = (a > b) ? a : b;
			}
			else
				K[i][w] = K[i -1][w];
			w++;
		}
		i++;
	}
	__syncthreads();*/
	//FOR LOOP
	for(int i =0;i <= (*n);i++) {
		for(int w = 0;w <= (*W);w++){
			
			if(i == 0 || w == 0)
				K[i][w] = 0;
			else if(wt[i -1] <= w) {
				int a,b;
				a = val[i-1] + K[i-1][w - wt[i - 1]];
				b = K[i-1][w];
				
				K[i][w] = (a > b) ? a : b;
			}
			else
				K[i][w] = K[i -1][w];
		}
	}
	*result = K[*n][*W];
}

int main() {
	int val[N] = {60,100,120};
	int wt[N] = {10,20,30};
	int W = GLOBAL_W;
	int result = 0;
	int n = N;
	
	int *dVal, *dWt, *dW, *dRes, *dN;
	
	//allocate GPU memory
	hipMalloc(&dVal,sizeof(val) * sizeof(int));
	hipMalloc(&dWt,sizeof(wt) * sizeof(int));
	hipMalloc(&dW,sizeof(int));
	hipMalloc(&dRes,sizeof(int));
	hipMalloc(&dN,sizeof(int));
	
	//copy values from cpu to gpu
	hipMemcpy(dVal, &val, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dWt, &wt, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dW, &W, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dRes, &result, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dW, &W, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dN, &n, sizeof(int), hipMemcpyHostToDevice);
	
	//call kernel function
	knapsack<<<BLOCKS,THREADS>>>(dW,dVal,dWt,dN,dRes);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy result from gpu to cpu
	hipMemcpy(&result,dRes, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Result: %d\n", result);
	
	//free gpu memory
	hipFree(dW);
	hipFree(dVal);
	hipFree(dWt);
	hipFree(dN);
	hipFree(dRes);
	return 0;
}
