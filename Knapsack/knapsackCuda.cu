#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKS 64
#define THREADS 1

#define N	3
#define GLOBAL_W	50

__global__ void knapsack(int *val, int *wt,int *result) {
	int K[N + 1][GLOBAL_W + 1];
	
	/*GPU parallel
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int w = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(i <= N && w <= GLOBAL_W) {			
		if(i == 0 || w == 0)
			K[i][w] = 0;
		else if(wt[i -1] <= w) {
			int a,b;
			a = val[i-1] + K[i-1][w - wt[i - 1]];
			b = K[i-1][w];
			
			K[i][w] = (a > b) ? a : b;
		}
		else
			K[i][w] = K[i -1][w];
	}
	__syncthreads();*/
	//FOR LOOP
	for(int i =0;i <= N;i++) {
		for(int w = 0;w <= GLOBAL_W;w++){
			
			if(i == 0 || w == 0)
				K[i][w] = 0;
			else if(wt[i -1] <= w) {
				int a,b;
				a = val[i-1] + K[i-1][w - wt[i - 1]];
				b = K[i-1][w];
				
				K[i][w] = (a > b) ? a : b;
			}
			else
				K[i][w] = K[i -1][w];
		}
	}
	*result = K[N][GLOBAL_W];
}

int main() {
	int val[N] = {60,100,120};
	int wt[N] = {10,20,30};
	int result = 0;
	
	//int W = GLOBAL_W;
	//int n = N;
	
	int *dVal, *dWt,*dRes;
	//int *dW,*dN;
	
	//allocate GPU memory
	hipMalloc(&dVal,sizeof(val) * sizeof(int));
	hipMalloc(&dWt,sizeof(wt) * sizeof(int));
	hipMalloc(&dRes,sizeof(int));
	
	//hipMalloc(&dW,sizeof(int));
	//hipMalloc(&dN,sizeof(int));
	
	//copy values from cpu to gpu
	hipMemcpy(dVal, &val, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dWt, &wt, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dRes, &result, sizeof(int), hipMemcpyHostToDevice);
	
	//hipMemcpy(dW, &W, sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(dN, &n, sizeof(int), hipMemcpyHostToDevice);
	
	//call kernel function
	knapsack<<<CORES,THREADS>>>(dVal,dWt,dRes);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy result from gpu to cpu
	hipMemcpy(&result,dRes, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Result: %d\n", result);
	
	//free gpu memory
	hipFree(dVal);
	hipFree(dWt);
	hipFree(dRes);
	//hipFree(dW);
	//hipFree(dN);
	return 0;
}
