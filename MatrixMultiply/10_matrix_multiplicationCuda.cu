#include "hip/hip_runtime.h"
//Matrix multiplication using shared and non shared kernal
#include "stdio.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#define TILE_SIZE 25
#define SIZE 100
//Matrix multiplication using non shared kernel
__global__ void gpu_Matrix_Mul_nonshared(float *d_a, float *d_b, float *d_c)
{
	int row, col;
	col = TILE_SIZE * blockIdx.x + threadIdx.x;
	row = TILE_SIZE * blockIdx.y + threadIdx.y;
	
	for (int k = 0; k< SIZE; k++)
	{
		d_c[row*SIZE + col] += d_a[row * SIZE + k] * d_b[k * SIZE + col];
	}
}

/* Matrix multiplication using shared kernel
__global__ void gpu_Matrix_Mul_shared(float *d_a, float *d_b, float *d_c, const int size)
{
	int row, col;
	//Defining Shared Memory
	__shared__ float shared_a[TILE_SIZE][TILE_SIZE];
	__shared__ float shared_b[TILE_SIZE][TILE_SIZE];
	col = TILE_SIZE * blockIdx.x + threadIdx.x;
	row = TILE_SIZE * blockIdx.y + threadIdx.y;

	for (int i = 0; i< size / TILE_SIZE; i++) 
	{
		shared_a[threadIdx.y][threadIdx.x] = d_a[row* size + (i*TILE_SIZE + threadIdx.x)];
		shared_b[threadIdx.y][threadIdx.x] = d_b[(i*TILE_SIZE + threadIdx.y) * size + col];
		__syncthreads(); 
		for (int j = 0; j<TILE_SIZE; j++)
			d_c[row*size + col] += shared_a[threadIdx.y][j] * shared_b[j][threadIdx.x];
		__syncthreads(); 

	}
}
*/
// main routine
int main()
{
	//Define Host Array
	float h_a[SIZE][SIZE], h_b[SIZE][SIZE],h_result[SIZE][SIZE];
	//Defining device Array
	float *d_a, *d_b, *d_result; 
	//Initialize host Array
	for (int i = 0; i<SIZE; i++)
	{
		for (int j = 0; j<SIZE; j++)
		{
			h_a[i][j] = i;
			h_b[i][j] = j;
		}
	}

	hipMalloc((void **)&d_a, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&d_b, SIZE*SIZE * sizeof(int));
	hipMalloc((void **)&d_result, SIZE*SIZE* sizeof(int));


	//copy host array to device array

	hipMemcpy(d_a, h_a, SIZE*SIZE* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, SIZE*SIZE* sizeof(int), hipMemcpyHostToDevice);
	
	//Define grid and block dimensions
	dim3 dimGrid(SIZE / TILE_SIZE, SIZE / TILE_SIZE, 1);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
	//gpu_Matrix_Mul_nonshared << <dimGrid, dimBlock >> > (d_a, d_b, d_result, size);

	gpu_Matrix_Mul_nonshared << <dimGrid, dimBlock >> > (d_a, d_b, d_result);

	hipMemcpy(h_result, d_result, SIZE*SIZE * sizeof(int),	hipMemcpyDeviceToHost);
	printf("The result of Matrix multiplication is: \n");
	
	for (int i = 0; i< SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%f   ", h_result[i][j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_result);
	return 0;
}
