//C program to multiply two square matrices.
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 4
#define BLOCKS 1
#define THREADS 1
// This function multiplies mat1[][] and mat2[][],
// and stores the result in res[][]
__global__ void multiply(int **mat1, int **mat2, int **res){
	/*int i = blockIdx.x * blockIdx.x + threadIdx.x;
	int j = blockIdx.y * blockIdx.y + threadIdx.y;
	int k = blockIdx.z * blockIdx.z + threadIdx.z;
	
	if(i < N) {
		if(j < N) {
			res[i][j] = 0;
			if(k < N)
				res[i][j] += mat1[i][k] * mat2[k][j];
		}
	}*/
	
	for(int i = 0;i < N;i++) {
		for(int j =0;j < N;j++) {
			res[i][j] = 0;
			for(int k =0;k < N;k++)
				res[i][j] += (mat1[i][k]) * (mat2[k][j]);
		}
	}
}
int main() {
	int **dMat1, **dMat2, **dRes;
	
	//allocate GPU memory
	hipMalloc(&dMat1, N * N * sizeof(int));
	hipMalloc(&dMat2, N * N * sizeof(int));
	hipMalloc(&dRes, N * N * sizeof(int));
	
	//CPU varibles
	int mat1[N][N] = { { 1, 1, 1, 1 },
					{ 2, 2, 2, 2 },
					{ 3, 3, 3, 3 },
					{ 4, 4, 4, 4 } };

	int mat2[N][N] = { { 1, 1, 1, 1 },
					{ 2, 2, 2, 2 },
					{ 3, 3, 3, 3 },
					{ 4, 4, 4, 4 } };
	int res[N][N]; // To store result
	
	
	//copy data from cpu to gpu
	hipMemcpy(&dMat1,&mat1, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(&dMat2,&mat2, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(&dRes,&res, N * N * sizeof(int), hipMemcpyHostToDevice);
	
	multiply<<<BLOCKS,THREADS>>>(dMat1,dMat2,dRes);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	//copy result data from GPU to CPU
	//cudaMemcpy(&mat1, &dMat1, N * N * sizeof(int), cudaMemcpyDeviceToHost);
	//cudaMemcpy(&mat2, &dMat2, N * N * sizeof(int), cudaMemcpyDeviceToHost);
	hipMemcpy(&res, &dRes, N * N * sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Result matrix is \n");
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
			printf("%d ", res[i][j]);
		printf("\n");
	}
	
	//free gpu memory
	hipFree(dMat1);
	hipFree(dMat2);
	hipFree(dRes);
	
	return 0;
}
