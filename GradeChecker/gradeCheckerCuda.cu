							
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define BLOCKS 1
#define THREADS 1
#define SIZE 1000
__global__ void gradeCheck(int *numGrad, char *gradLetter) {
	int tid = threadIdx.x;
	
	if(numGrad[tid] >= 90 && numGrad[tid] <= 100)
		gradLetter[tid] = 'A';
	
	else if(numGrad[tid] >= 80 && numGrad[tid] < 90)
		gradLetter[tid] = 'B';
	
	else if(numGrad[tid] >= 70 && numGrad[tid] < 80)
		gradLetter[tid] = 'C';
	
	else if(numGrad[tid] >= 60 && numGrad[tid] < 70)
		gradLetter[tid] = 'D';
	
	else if(numGrad[tid] >= 0 && numGrad[tid] < 60)
		gradLetter[tid] = 'F';
		
		__syncthreads();
		
}
int main() {
	int marks[SIZE];
	char letter[SIZE];
	
	int *dMarks;
	char *dLetter;
	
	int upper = 100;
	int lower = 50;
	
	//Initialize  Array
	for (int i = 0; i<SIZE; i++){
		marks[i] = (rand() % (upper - lower + 1)) + lower;
		letter[i] = 'F';
	}
	
	//allocate memory
	hipMalloc(&dMarks,SIZE * sizeof(int));
	hipMalloc(&dLetter,SIZE * sizeof(char));
	
	//copy data from cpu to gpu
	hipMemcpy(dMarks,&marks,SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dLetter,&letter,SIZE * sizeof(char), hipMemcpyHostToDevice);
	
	//execute kernel function
	gradeCheck<<<BLOCKS,SIZE>>>(dMarks,dLetter);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy data from gpu to cpu
	hipMemcpy(&letter,dLetter, SIZE * sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(&marks,dMarks, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	
	//print result
	for(int i =0; i<SIZE;i++)
		printf("Number Grade: %d\t Letter Grade: %c\n",marks[i],letter[i]);
	
	//free gpu memory
	hipFree(dMarks);
	hipFree(dLetter);
	return 0;
}
