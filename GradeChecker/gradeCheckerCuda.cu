							
#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKS 1
#define THREADS 1
__global__ void gradeCheck(int *numGrad, char *gradLetter) {
	
	if(*numGrad >= 90 && *numGrad <= 100)
		*gradLetter = 'A';
		
	else if(*numGrad >= 80 && *numGrad < 90)
		*gradLetter = 'B';
		
	else if(*numGrad >= 70 && *numGrad < 80)
		*gradLetter = 'C';
		
	else if(*numGrad >= 60 && *numGrad < 70)
		*gradLetter = 'D';
		
	else if(*numGrad >= 0 && *numGrad < 60)
		*gradLetter = 'F';
		
}
int main() {
	int mark = 90;
	char letter = 'F';
	
	int *dMark;
	char *dLetter;
	
	//allocate memory
	hipMalloc(&dMark,sizeof(int));
	hipMalloc(&dLetter,sizeof(char));
	
	//copy data from cpu to gpu
	hipMemcpy(dMark,&mark,sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dLetter,&letter,sizeof(char), hipMemcpyHostToDevice);
	
	//execute kernel function
	gradeCheck<<<BLOCKS,THREADS>>>(dMark,dLetter);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy data from gpu to cpu
	hipMemcpy(&letter,dLetter, sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(&mark,dMark, sizeof(int), hipMemcpyDeviceToHost);
	
	//print result
	printf("Number Grade: %d\t Letter Grade: %c\n",mark,letter);
	
	//free gpu memory
	hipFree(dMark);
	hipFree(dLetter);
	return 0;
}
