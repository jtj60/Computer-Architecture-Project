							
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define BLOCKS 1
#define THREADS 1
#define SIZE 10000
#define TILE_SIZE 32
__global__ void gradeCheck(int *numGrad, char *gradLetter) {
	int blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
	int tid = (blockId * blockDim.x) + threadIdx.x;
	
	if(numGrad[tid] >= 90 && numGrad[tid] <= 100)
		gradLetter[tid] = 'A';
	
	else if(numGrad[tid] >= 80 && numGrad[tid] < 90)
		gradLetter[tid] = 'B';
	
	else if(numGrad[tid] >= 70 && numGrad[tid] < 80)
		gradLetter[tid] = 'C';
	
	else if(numGrad[tid] >= 60 && numGrad[tid] < 70)
		gradLetter[tid] = 'D';
	
	else if(numGrad[tid] >= 0 && numGrad[tid] < 60)
		gradLetter[tid] = 'F';
		
	//printf("ThreadId: %d: Number Grade: %d\t Letter Grade: %c\n",tid,numGrad[tid],gradLetter[tid]);
		__syncthreads();
		
}
//__global__ void printArray(int *numGrad, char *gradLetter)
int main() {
	int marks[SIZE];
	char letter[SIZE];
	
	int *dMarks;
	char *dLetter;
	
	int upper = 100;
	int lower = 50;
	
	//Initialize  Array
	for (int i = 0; i<SIZE; i++){
		marks[i] = (rand() % (upper - lower + 1)) + lower;
		letter[i] = 'F';
	}
	
	//allocate memory
	hipMalloc(&dMarks,SIZE * sizeof(int));
	hipMalloc(&dLetter,SIZE * sizeof(char));
	
	//copy data from cpu to gpu
	hipMemcpy(dMarks,&marks,SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dLetter,&letter,SIZE * sizeof(char), hipMemcpyHostToDevice);
	
	//Define grid and block dimensions
	dim3 dimGrid(SIZE/100,SIZE/100,1);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
	
	//execute kernel function
	gradeCheck<<<dimGrid,dimBlock>>>(dMarks,dLetter);
	
	//wait for gpu to finish
	hipDeviceSynchronize();
	
	//copy data from gpu to cpu
	hipMemcpy(&letter,dLetter, SIZE * sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(&marks,dMarks, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	
	//print result
	for(int i =0; i<SIZE;i++)
		printf("%d: Number Grade: %d\t Letter Grade: %c\n",i,marks[i],letter[i]);
	
	//free gpu memory
	hipFree(dMarks);
	hipFree(dLetter);
	return 0;
}
